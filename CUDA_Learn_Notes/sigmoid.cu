
#include <hip/hip_runtime.h>



#define FLOAT4(value) (reinterpret_cast<float4*>(&(value))[0])
// Sigmoid x: N, y: N y=1/(1+exp(-x))
// grid(N/128), block(K=128) 
__global__ void sigmoid(float* x, float* y, int N) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < N) y[idx] = 1.0f / (1.0f + expf(-x[idx]));
}

// Sigmoid x: N, y: N y=1/(1+exp(-x)) Vec4
// grid(N/128), block(128/4)
__global__ void sigmoid_vec4(float* x, float* y, int N) {
  int idx = (blockIdx.x * blockDim.x + threadIdx.x) * 4;
  if (idx < N) {
    float4 reg_x = FLOAT4(x[idx]);
    float4 reg_y;
    reg_y.x = 1.0f / (1.0f + expf(-reg_x.x));
    reg_y.y = 1.0f / (1.0f + expf(-reg_x.y));
    reg_y.z = 1.0f / (1.0f + expf(-reg_x.z));
    reg_y.w = 1.0f / (1.0f + expf(-reg_x.w));
    FLOAT4(y[idx]) = reg_y;
  }
}