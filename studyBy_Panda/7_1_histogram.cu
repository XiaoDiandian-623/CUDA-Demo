#include <bits/stdc++.h>
#include <hip/hip_runtime.h>

//2.389ms
//tips: L68��L75Ϊ����������bug��˵�����ڱ�����ʱ����Ҫ��ȷ���������������˻����˶����ܻ��������ֵ����cuda�ɴ���������������
template <int blockSize>
__global__ void histgram(int *hist_data, int *bin_data, int N)
{
    __shared__ int cache[256];
    int gtid = blockIdx.x * blockSize + threadIdx.x; // ��ָ��ǰ�߳�������block��Χ�ڵ�ȫ��id
    int tid = threadIdx.x; // ��ָ��ǰ�߳�����block�ڵ�id
    cache[tid] = 0; // ÿ��thread��ʼ��shared mem
    __syncthreads();
    // forѭ�����Զ�ȷ��ÿ���̴߳����Ԫ�ظ���
    for (int i = gtid; i < N; i += gridDim.x * blockSize)
    {
        int val = hist_data[i];// ÿ�����̼߳���ȫ���ڴ��е����ɸ�ֵ
        atomicAdd(&cache[val], 1); // ԭ�Ӽӷ���ǿ��ʹ�ò��е�CUDA�̴߳���ִ�мӷ������ǲ����ܱ�֤˳��
    }
    __syncthreads();//�˿�ÿ��block��bin����ͳ����cache���smem��
    //debug info: if(tid== 0){printf("cache[1]=%d,hist[1]=%d\n",cache[1],hist_data[2]);}
    atomicAdd(&bin_data[tid], cache[tid]);
    //debug info: if(tid== 0){printf("bin_data[1]=%d,hist[1]=%d\n",bin_data[1],hist_data[2]);}
}

bool CheckResult(int *out, int* groudtruth, int N){
    for (int i = 0; i < N; i++){
        if (out[i] != groudtruth[i]) {
            printf("in checkres, out[i]=%d, gt[i]=%d\n", out[i], groudtruth[i]);
            return false;
        }
    }
    return true;
}

int main(){
    float milliseconds = 0;
    const int N = 25600000;
    int *hist = (int *)malloc(N * sizeof(int));
    int *bin = (int *)malloc(256 * sizeof(int));
    int *bin_data;
    int *hist_data;
    hipMalloc((void **)&bin_data, 256 * sizeof(int));
    hipMalloc((void **)&hist_data, N * sizeof(int));

    for(int i = 0; i < N; i++){
        hist[i] = i % 256;
    }

    int *groudtruth = (int *)malloc(256 * sizeof(int));;
    for(int j = 0; j < 256; j++){
        groudtruth[j] = 100000;
    }

    hipMemcpy(hist_data, hist, N * sizeof(int), hipMemcpyHostToDevice);
    hipSetDevice(0);
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);
    const int blockSize = 256;
    int GridSize = std::min((N + 256 - 1) / 256, deviceProp.maxGridSize[0]);
    dim3 Grid(GridSize);
    dim3 Block(blockSize);
    
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    // bug1: L68��N���ܴ���֮ǰ����256������L19��cache[1]��ӡ����Ϊ0
    histgram<blockSize><<<Grid, Block>>>(hist_data, bin_data, N);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);

    hipMemcpy(bin, bin_data, 256 * sizeof(int), hipMemcpyDeviceToHost);
    // bug2: ͬbug1��L67����ȥ��256��ʾ����buffer�����������������þ�ȷ��֮ǰ����N������ֻ��ӡ��1��ֵ������Ȼ����L27��ӡ������ֵΪ����ֵ
    bool is_right = CheckResult(bin, groudtruth, 256);
    if(is_right) {
        printf("the ans is right\n");
    } else {
        printf("the ans is wrong\n");
        for(int i = 0; i < 256; i++){
            printf("%d ", bin[i]);
        }
        printf("\n");
    }
    printf("histogram + shared_mem + multi_value latency = %f ms\n", milliseconds);    

    hipFree(bin_data);
    hipFree(hist_data);
    free(bin);
    free(hist);
}