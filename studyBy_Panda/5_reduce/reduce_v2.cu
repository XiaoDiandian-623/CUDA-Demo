#include<bits/stdc++.h>
#include<hip/hip_runtime.h>


//消除shared memory 的 bank conflict

template<int blockSize>
__global__ void reduce_v2(float* d_in,float* d_out) 
{
    __shared__ float smem[blockSize];

    unsigned int tid = threadIdx.x;//当前线程在所有block范围内的全局id
    unsigned int gtid = blockIdx.x * blockSize + threadIdx.x;

    smem[tid] = d_in[gtid] ;
    __syncthreads();


    //并行部分
    for(unsigned int index=blockDim.x/2;index>0;index>>=1) {
        if(tid < index) {
            smem[tid] += smem[tid+index];
        }
        __syncthreads();
    }

    if(tid==0) {
        d_out[blockIdx.x] = smem[0];
    }

}

bool CheckResult(float* out,float groudtruth,int n) 
{
    int res = 0;
    for(int i=0;i<n;i++) {
        res += out[i];
    }
    if(res != groudtruth) {
        return false;
    }
    return true;
}

int main()
{
    float milliseconds = 0;
    const int N = 25600000;

    hipSetDevice(0);
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp,0);

    const int blockSize = 256;
    int GridSize = std::min((N+blockSize-1)/blockSize,deviceProp.maxGridSize[0]);
     double msecPerMatrixMul[2] = {0,0};
    double gigaFlops[2] = {0,0};
    double flopPerMatrixMul = 2.0 * N;
    float* a = (float*)malloc(N*sizeof(float));
    float* d_a;
    hipMalloc((void**)&d_a,N*sizeof(float));

    float* out = (float*)malloc(GridSize*sizeof(float));
    float* d_out;
    hipMalloc((void**)&d_out,GridSize*sizeof(float));

    for(int i=0;i<N;i++) {
        a[i]=2.0f;
    }
    float groudtruth = N*2.0f;

    hipMemcpy(d_a,a,N*sizeof(float),hipMemcpyHostToDevice);

    dim3 Grid(GridSize);
    dim3 Block(blockSize);

    hipEvent_t start,stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    reduce_v2<blockSize><<<Grid,Block>>>(d_a,d_out);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds,start,stop);

    hipMemcpy(out,d_out,GridSize*sizeof(float),hipMemcpyDeviceToHost);
    printf("allcated %d blocks,data counts %d\n",GridSize,N);

    bool is_right = CheckResult(out,groudtruth,GridSize);
    if(is_right) {
        printf("the ans is right\n");
    }else {
        printf("the ans is wrong\n");

        printf("the groudtruth is %f\n",groudtruth);
    }
msecPerMatrixMul[0] = milliseconds;
    gigaFlops[0] = (flopPerMatrixMul * 1.0e-9f) / (msecPerMatrixMul[0] / 1000.0f);
    printf("Gemm performance = %.2f GFlop/s,Time= %.3f,Size = %.0f Ops,\n",gigaFlops[0],msecPerMatrixMul[0],flopPerMatrixMul);
    printf("reduce_v3 latency = %f ms\n", milliseconds);
    printf("the reduce_v2 latency is %f ms\n",milliseconds);

    hipFree(d_a);
    hipFree(d_out);
    free(a);
    free(out);



}
