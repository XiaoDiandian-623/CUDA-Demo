#include <bits/stdc++.h>
#include <hip/hip_runtime.h>


// v1: ����ǰ���ε�warp divergence �߳����ֻ�

// blockSize��Ϊģ�������Ч����Ҫ���ھ�̬shared memory��������Ҫ��������ڳ���ָ����С��L10)
template<int blockSize>
__global__ void reduce_v1(float *d_in,float *d_out){
    __shared__ float smem[blockSize];

    unsigned int tid = threadIdx.x;
    
    unsigned int gtid = blockIdx.x * blockSize + threadIdx.x;
   
    smem[tid] = d_in[gtid];
   
    __syncthreads();

    // ����v0�����Ľ���ʹ��һ��warp�ڵ��̲߳��ᱻL23���ֻ���������ִ�в��ֵȴ�
    // ��ʱһ��block��d_in������ݵ�reduce sum���������idΪ0���߳�����
    for(unsigned int s = 1; s < blockDim.x; s *= 2) {
        int index = 2 * s * tid;
        if (index < blockDim.x) {
            smem[index] += smem[index + s];
        }
        __syncthreads();
    }

   
    if (tid == 0) {
        d_out[blockIdx.x] = smem[0];
    }
}

bool CheckResult(float *out, float groudtruth, int n){
    float res = 0;
    for (int i = 0; i < n; i++){
        res += out[i];
    }
    if (res != groudtruth) {
        return false;
    }
    return true;
}

int main(){
    float milliseconds = 0;
    //const int N = 32 * 1024 * 1024;
    const int N = 25600000;
    hipSetDevice(0);
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);
    const int blockSize = 256;
    int GridSize = std::min((N + 256 - 1) / 256, deviceProp.maxGridSize[0]);
     double msecPerMatrixMul[2] = {0,0};
    double gigaFlops[2] = {0,0};
    double flopPerMatrixMul = 2.0 * N;
    //int GridSize = 100000;
    float* a = (float *)malloc(N * sizeof(float));
    float* d_a;
    hipMalloc((void **)&d_a, N * sizeof(float));

    float* out = (float*)malloc((GridSize) * sizeof(float));
    float* d_out;
    hipMalloc((void **)&d_out, (GridSize) * sizeof(float));

    for(int i = 0; i < N; i++){
        a[i] = 1.0f;
    }

    float groudtruth = N * 1.0f;

    hipMemcpy(d_a, a, N * sizeof(float), hipMemcpyHostToDevice);

    dim3 Grid(GridSize);
    dim3 Block(blockSize);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    reduce_v1<blockSize><<<Grid,Block>>>(d_a, d_out);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);

    hipMemcpy(out, d_out, GridSize * sizeof(float), hipMemcpyDeviceToHost);
    printf("allcated %d blocks, data counts are %d", GridSize, N);
    bool is_right = CheckResult(out, groudtruth, GridSize);
    if(is_right) {
        printf("the ans is right\n");
    } else {
        printf("the ans is wrong\n");
        //for(int i = 0; i < GridSize;i++){
            //printf("res per block : %lf ",out[i]);
        //}
        //printf("\n");
        printf("groudtruth is: %f \n", groudtruth);
    }
    msecPerMatrixMul[0] = milliseconds;
    gigaFlops[0] = (flopPerMatrixMul * 1.0e-9f) / (msecPerMatrixMul[0] / 1000.0f);
    printf("Gemm performance = %.2f GFlop/s,Time= %.3f,Size = %.0f Ops,\n",gigaFlops[0],msecPerMatrixMul[0],flopPerMatrixMul);
    printf("reduce_v1 latency = %f ms\n", milliseconds);

    hipFree(d_a);
    hipFree(d_out);
    free(a);
    free(out);
}