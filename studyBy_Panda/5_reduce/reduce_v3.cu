#include <bits/stdc++.h>
#include <hip/hip_runtime.h>


//v3: 让空闲线程也干活

template<int blockSize>
__global__ void reduce_v3(float *d_in, float *d_out){
    __shared__ float smem[blockSize];
    
    unsigned int tid = threadIdx.x;
    // 泛指当前线程在所有block范围内的全局id, *2代表当前block要处理2*blocksize的数据
    // ep. blocksize = 2, blockIdx.x = 1, when tid = 0, gtid = 4, gtid + blockSize = 6; when tid = 1, gtid = 5, gtid + blockSize = 7
    // ep. blocksize = 2, blockIdx.x = 0, when tid = 0, gtid = 0, gtid + blockSize = 2; when tid = 1, gtid = 1, gtid + blockSize = 3
  
    unsigned int gtid = blockIdx.x * (blockSize * 2) + threadIdx.x;

    smem[tid] = d_in[gtid] + d_in[gtid + blockSize];
    __syncthreads();

   
    for (unsigned int index = blockDim.x / 2; index > 0; index >>= 1) {
        if (tid < index) {
            smem[tid] += smem[tid + index];
        }
        __syncthreads();
    }

    // 把reduce结果写回显存
    if (tid == 0) {
        d_out[blockIdx.x] = smem[0];
    }
}

bool CheckResult(float *out, float groudtruth, int n){
    float res = 0;
    for (int i = 0; i < n; i++){
        res += out[i];
    }
    if (res != groudtruth) {
        return false;
    }
    return true;
}

int main(){
    float milliseconds = 0;
    //const int N = 32 * 1024 * 1024;
    const int N = 25600000;
    hipSetDevice(0);
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);
    const int blockSize = 256;
    int GridSize = std::min((N + 256 - 1) / 256, deviceProp.maxGridSize[0]);
    double msecPerMatrixMul[2] = {0,0};
    double gigaFlops[2] = {0,0};
    double flopPerMatrixMul = 2.0 * N;
    //int GridSize = 100000;
    float *a = (float *)malloc(N * sizeof(float));
    float *d_a;
    hipMalloc((void **)&d_a, N * sizeof(float));

    float *out = (float*)malloc((GridSize) * sizeof(float));
    float *d_out;
    hipMalloc((void **)&d_out, (GridSize) * sizeof(float));

    for(int i = 0; i < N; i++){
        a[i] = 1.0f;
    }

    float groudtruth = N * 1.0f;

    hipMemcpy(d_a, a, N * sizeof(float), hipMemcpyHostToDevice);

    dim3 Grid(GridSize);
    dim3 Block(blockSize / 2);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    reduce_v3<blockSize / 2><<<Grid,Block>>>(d_a, d_out);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);

    hipMemcpy(out, d_out, GridSize * sizeof(float), hipMemcpyDeviceToHost);
    printf("allcated %d blocks, data counts are %d", GridSize, N);
    bool is_right = CheckResult(out, groudtruth, GridSize);
    if(is_right) {
        printf("the ans is right\n");
    } else {
        printf("the ans is wrong\n");
        
        printf("groudtruth is: %f \n", groudtruth);
    }
     msecPerMatrixMul[0] = milliseconds;
    gigaFlops[0] = (flopPerMatrixMul * 1.0e-9f) / (msecPerMatrixMul[0] / 1000.0f);
    printf("Gemm performance = %.2f GFlop/s,Time= %.3f,Size = %.0f Ops,\n",gigaFlops[0],msecPerMatrixMul[0],flopPerMatrixMul);
    printf("reduce_v3 latency = %f ms\n", milliseconds);

    hipFree(d_a);
    hipFree(d_out);
    free(a);
    free(out);
}