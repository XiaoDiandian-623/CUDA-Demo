#include <bits/stdc++.h>
#include <hip/hip_runtime.h>


#define THREAD_PER_BLOCK 256

// v5：for循环展开 省掉判断和加法指令
template <int blockSize>
__device__ void BlockSharedMemReduce(float* smem) {
    //对v4 的for循环展开，以减去for循环中的加法指令，以及给编译器更多重排指令的空间
  if (blockSize >= 1024) {
    if (threadIdx.x < 512) {
      smem[threadIdx.x] += smem[threadIdx.x + 512];
    }
    __syncthreads();
  }
  if (blockSize >= 512) {
    if (threadIdx.x < 256) {
      smem[threadIdx.x] += smem[threadIdx.x + 256];
    }
    __syncthreads();
  }
  if (blockSize >= 256) {
    if (threadIdx.x < 128) {
      smem[threadIdx.x] += smem[threadIdx.x + 128];
    }
    __syncthreads();
  }
  if (blockSize >= 128) {
    if (threadIdx.x < 64) {
      smem[threadIdx.x] += smem[threadIdx.x + 64];
    }
    __syncthreads();
  }
  // the final warp
  if (threadIdx.x < 32) {
    volatile float* vshm = smem;
    if (blockDim.x >= 64) {
      vshm[threadIdx.x] += vshm[threadIdx.x + 32];
    }
    vshm[threadIdx.x] += vshm[threadIdx.x + 16];
    vshm[threadIdx.x] += vshm[threadIdx.x + 8];
    vshm[threadIdx.x] += vshm[threadIdx.x + 4];
    vshm[threadIdx.x] += vshm[threadIdx.x + 2]; 
    vshm[threadIdx.x] += vshm[threadIdx.x + 1];
  }
}

template <int blockSize>
__global__ void reduce_v5(float *d_in, float *d_out){
    __shared__ float smem[THREAD_PER_BLOCK];
    // 泛指当前线程在其block内的id
    unsigned int tid = threadIdx.x;
    // 泛指当前线程在所有block范围内的全局id, *2代表当前block要处理2*blocksize的数据
    // ep. blocksize = 2, blockIdx.x = 1, when tid = 0, gtid = 4, gtid + blockSize = 6; when tid = 1, gtid = 5, gtid + blockSize = 7
    // ep. blocksize = 2, blockIdx.x = 0, when tid = 0, gtid = 0, gtid + blockSize = 2; when tid = 1, gtid = 1, gtid + blockSize = 3
    // so, we can understand L59, one thread handle data located in tid and tid + blockSize 
    unsigned int i = blockIdx.x * (blockDim.x * 2) + threadIdx.x;
    // load: 每个线程加载两个元素到shared mem对应位置
    smem[tid] = d_in[i] + d_in[i + blockDim.x];
    __syncthreads();
    // compute: reduce in shared mem
    BlockSharedMemReduce<blockSize>(smem);

    // store: 哪里来回哪里去，把reduce结果写回显存
    if (tid == 0) {
        d_out[blockIdx.x] = smem[0];
    }
}

bool CheckResult(float *out, float groudtruth, int n){
    float res = 0;
    for (int i = 0; i < n; i++){
        res += out[i];
    }
    if (res != groudtruth) {
        return false;
    }
    return true;
}

int main(){
    float milliseconds = 0;
    
    const int N = 25600000;
    hipSetDevice(0);
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);
    const int blockSize = 256;
    int GridSize = std::min((N + 256 - 1) / 256, deviceProp.maxGridSize[0]);
    //int GridSize = 100000;
    float *a = (float *)malloc(N * sizeof(float));
    float *d_a;
    hipMalloc((void **)&d_a, N * sizeof(float));

    float *out = (float*)malloc((GridSize) * sizeof(float));
    float *d_out;
    hipMalloc((void **)&d_out, (GridSize) * sizeof(float));

    for(int i = 0; i < N; i++){
        a[i] = 1.0f;
    }

    float groudtruth = N * 1.0f;

    hipMemcpy(d_a, a, N * sizeof(float), hipMemcpyHostToDevice);

    dim3 Grid(GridSize);
    dim3 Block(blockSize / 2);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    reduce_v5<blockSize / 2><<<Grid,Block>>>(d_a, d_out);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);

    hipMemcpy(out, d_out, GridSize * sizeof(float), hipMemcpyDeviceToHost);
    printf("allcated %d blocks, data counts are %d \n", GridSize, N);
    bool is_right = CheckResult(out, groudtruth, GridSize);
    if(is_right) {
        printf("the ans is right\n");
    } else {
        printf("the ans is wrong\n");
        for(int i = 0; i < GridSize;i++){
            printf("resPerBlock : %lf ",out[i]);
        }
        printf("\n");
        printf("groudtruth is: %f \n", groudtruth);
    }
    printf("reduce_v5 latency = %f ms\n", milliseconds);

    hipFree(d_a);
    hipFree(d_out);
    free(a);
    free(out);
}