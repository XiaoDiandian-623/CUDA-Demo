#include <bits/stdc++.h>
#include <hip/hip_runtime.h>



__global__ void reduce_baseline(const int* input, int* output, size_t n) {
    int sum = 0;
    for (size_t i = 0; i < n; ++i) {
        sum += input[i];
    }    
    *output = sum;
}

bool CheckResult(int *out, int groudtruth, int n){
    if (*out != groudtruth) {
        return false;
    }
    return true;
}

int main(){
    float milliseconds = 0;
    //const int N = 32 * 1024 * 1024;
    const int N = 25600000;
    hipSetDevice(0);
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);
    const int blockSize = 256;
    //const int blockSize = 1;
    int GridSize = std::min((N + 256 - 1) / 256, deviceProp.maxGridSize[0]);//used later
    //int GridSize = 1;
double msecPerMatrixMul[2] = {0,0};
    double gigaFlops[2] = {0,0};
    double flopPerMatrixMul = 2.0 * N;
    // �����ڴ���Դ沢��ʼ������
    int *a = (int *)malloc(N * sizeof(int));
    int *d_a;
    hipMalloc((void **)&d_a, N * sizeof(int));

    int *out = (int*)malloc((GridSize) * sizeof(int));
    int *d_out;
    hipMalloc((void **)&d_out, (GridSize) * sizeof(int));

    for(int i = 0; i < N; i++){
        a[i] = 1;
    }

    int groudtruth = N * 1;
    // �ѳ�ʼ��������ݿ�����GPU
    hipMemcpy(d_a, a, N * sizeof(int), hipMemcpyHostToDevice);
    // ��������block������threads����
    dim3 Grid(GridSize);
    dim3 Block(blockSize);

    //��ʱ
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    // ����1��block��1��thread
    reduce_baseline<<<1, 1>>>(d_a, d_out, N);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);


    // ���������CPU��check��ȷ��
    hipMemcpy(out, d_out, GridSize * sizeof(int), hipMemcpyDeviceToHost);
    printf("allcated %d blocks, data counts are %d\n", GridSize, N);
    bool is_right = CheckResult(out, groudtruth, GridSize);
    if(is_right) {
        printf("the ans is right\n");
    } else {
        printf("the ans is wrong\n");
        for(int i = 0; i < GridSize;i++){
            printf("res per block : %lf ",out[i]);
        }
        printf("\n");
        printf("groudtruth is: %f \n", groudtruth);
    }
    msecPerMatrixMul[0] = milliseconds;
    gigaFlops[0] = (flopPerMatrixMul * 1.0e-9f) / (msecPerMatrixMul[0] / 1000.0f);
    printf("Gemm performance = %.2f GFlop/s,Time= %.3f,Size = %.0f Ops,\n",gigaFlops[0],msecPerMatrixMul[0],flopPerMatrixMul);
    printf("result: %f",out);
    printf("reduce_baseline latency = %f ms\n", milliseconds);


    hipFree(d_a);
    hipFree(d_out);
    free(a);
    free(out);
}