#include <bits/stdc++.h>
#include <hip/hip_runtime.h>


// v6: multi-block reduce final result by two pass
// latency: 1.817248ms
template <int blockSize>
__device__ void BlockSharedMemReduce(float* smem) {
    //对v4的for循环展开，以减去for循环中的加法指令，以及给编译器更多重排指令的空间
  if (blockSize >= 1024) {
    if (threadIdx.x < 512) {
      smem[threadIdx.x] += smem[threadIdx.x + 512];
    }
    __syncthreads();
  }
  if (blockSize >= 512) {
    if (threadIdx.x < 256) {
      smem[threadIdx.x] += smem[threadIdx.x + 256];
    }
    __syncthreads();
  }
  if (blockSize >= 256) {
    if (threadIdx.x < 128) {
      smem[threadIdx.x] += smem[threadIdx.x + 128];
    }
    __syncthreads();
  }
  if (blockSize >= 128) {
    if (threadIdx.x < 64) {
      smem[threadIdx.x] += smem[threadIdx.x + 64];
    }
    __syncthreads();
  }
  // the final warp 展开避免了执行循环控制和线程同步逻辑
  if (threadIdx.x < 32) {
    volatile float* vshm = smem;
    if (blockDim.x >= 64) {
      vshm[threadIdx.x] += vshm[threadIdx.x + 32];
    }
    vshm[threadIdx.x] += vshm[threadIdx.x + 16];
    vshm[threadIdx.x] += vshm[threadIdx.x + 8];
    vshm[threadIdx.x] += vshm[threadIdx.x + 4];
    vshm[threadIdx.x] += vshm[threadIdx.x + 2];                                                                                                                                                                                          vshm[threadIdx.x] += vshm[threadIdx.x + 1];

  }
}

template <int blockSize>
__global__ void reduce_v6(float *d_in, float *d_out, int nums){
    __shared__ float smem[blockSize];
    
    unsigned int tid = threadIdx.x;
    
    unsigned int gtid = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int total_thread_num = blockDim.x * gridDim.x;
    // 基于v5的改进：不用显式指定一个线程处理2个元素，而是通过L58的for循环来自动确定每个线程处理的元素个数
    float sum = 0.0f;
    for (int32_t i = gtid; i < nums; i += total_thread_num) {
        sum += d_in[i];
    }
    smem[tid] = sum;
    __syncthreads();
    // compute: reduce in shared mem
    BlockSharedMemReduce<blockSize>(smem);

    if (tid == 0) {
        d_out[blockIdx.x] = smem[0];
    }
}

bool CheckResult(float *out, float groudtruth, int n){
    if (*out != groudtruth) {
      return false;
    }
    return true;
}

int main(){
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);
    int maxblocks = deviceProp.maxGridSize[0];
    const int blockSize = 256;
    const int N = 25600000;
    int gridSize = std::min((N + blockSize - 1) / blockSize, maxblocks);

    float milliseconds = 0;
    float *a = (float *)malloc(N * sizeof(float));
    float *d_a;
    hipMalloc((void **)&d_a,N * sizeof(float));

    float *out = (float*)malloc((gridSize) * sizeof(float));
    float *d_out;
    float *part_out;//新增part_out存储每个block reduce的结果
    hipMalloc((void **)&d_out, 1 * sizeof(float));
    hipMalloc((void **)&part_out, (gridSize) * sizeof(float));
    float groudtruth = N;

    double msecPerMatrixMul[2] = {0,0};
    double gigaFlops[2] = {0,0};
    double flopPerMatrixMul = 2.0 * N;
    
    for(int i = 0; i < N; i++){
        a[i] = 1;
    }

    hipMemcpy(d_a, a, N * sizeof(float), hipMemcpyHostToDevice);

    dim3 Grid(gridSize);
    dim3 Block(blockSize);
    

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    reduce_v6<blockSize><<<Grid, Block>>>(d_a, part_out, N);
    reduce_v6<blockSize><<<1, Block>>>(part_out, d_out, gridSize);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);

    hipMemcpy(out, d_out, 1 * sizeof(float), hipMemcpyDeviceToHost);
    bool is_right = CheckResult(out, groudtruth, 1);
    if(is_right) {
        printf("the ans is right\n");
    } else {
        printf("the ans is wrong\n");
        for(int i = 0;i < 1;i++){
            printf("%lf ",out[i]);
        }
        printf("\n");
    }

    msecPerMatrixMul[0] = milliseconds;
    gigaFlops[0] = (flopPerMatrixMul * 1.0e-9f) / (msecPerMatrixMul[0] / 1000.0f);
    printf("Gemm performance = %.2f GFlop/s,Time= %.3f,Size = %.0f Ops,\n",gigaFlops[0],msecPerMatrixMul[0],flopPerMatrixMul);
    printf("reduce_v6 latency = %f ms\n", milliseconds);

    hipFree(d_a);
    hipFree(d_out);
    hipFree(part_out);
    free(a);
    free(out);
}