#include<bits/stdc++.h>
#include<hip/hip_runtime.h>


template<int blockSize>
__global__ void reduce_v0(float* d_in,float* d_out)
{
    __shared__ float smem[blockSize];

    int tid = threadIdx.x;
    int gtid = blockIdx.x * blockSize + threadIdx.x;

    smem[tid] = d_in[gtid];
    __syncthreads();

    //���в���
    for(int index=1;index<blockDim.x;index *=2) {
        if(tid %(2*index) == 0) {
            smem[tid] += smem[tid+index];
        }
    __syncthreads();
    }

    if(tid==0) {
        d_out[blockIdx.x] = smem[0];
    }

}

bool checkResult(float* out,float groudtruth,int n) 
{
   float res = 0;
   for(int i=0;i<n;i++) {
    res += out[i];
   } 
   if(res!=groudtruth) {
    return false;
   }
   return true;
}

int main()
{
    float milliseconds = 0;
    const int N = 25600000;

    hipSetDevice(0);
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp,0);

    const int blockSize = 256;//ÿ��block�д���256��thread
    int GridSize = std::min((N+256-1)/256,deviceProp.maxGridSize[0]);
 double msecPerMatrixMul[2] = {0,0};
    double gigaFlops[2] = {0,0};
    double flopPerMatrixMul = 2.0 * N;
    float* a = (float*)malloc(N*sizeof(float));
    float* d_a;
    hipMalloc((void**)&d_a,N*sizeof(float));//����ָ��

    float* out = (float*)malloc(N*sizeof(float));
    float* d_out;
    hipMalloc((void**)&d_out,N*sizeof(float));

    for(int i=0;i<N;i++) {
        a[i] = 1.0f;
    }

    float groudtruth = N*1.0f;

    hipMemcpy(d_a,a,N*sizeof(float),hipMemcpyHostToDevice);

    dim3 Grid(GridSize);
    dim3 Block(blockSize);

    //��ʱAPI
    hipEvent_t start,stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    reduce_v0<blockSize><<<Grid,Block>>>(d_a,d_out);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds,start,stop);

    hipMemcpy(out,d_out,GridSize*sizeof(float),hipMemcpyDeviceToHost);

    printf("allcated %d blocks,data counts are %d",GridSize,N);
    bool is_right = checkResult(out,groudtruth,GridSize);
    if(is_right) {
        printf("the ans is right\n");
    }else {
        printf("the ans is wrong\n");
        printf("groudtruth is: %f ms\n",milliseconds);
    }
msecPerMatrixMul[0] = milliseconds;
    gigaFlops[0] = (flopPerMatrixMul * 1.0e-9f) / (msecPerMatrixMul[0] / 1000.0f);
    printf("Gemm performance = %.2f GFlop/s,Time= %.3f,Size = %.0f Ops,\n",gigaFlops[0],msecPerMatrixMul[0],flopPerMatrixMul);
    printf("reduce_v0 latency = %f ms\n",milliseconds);

    hipFree(d_a);
    hipFree(d_out);
    free(a);
    free(out);


}